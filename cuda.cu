#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"


__device__ int isGolomb(int *x, int in) {
	int map[100];
	for (int i = 0; i<100; i++) map[i] = 0;

	for (int i = 0; i<in; i++) {
		for (int j = i + 1; j<in; j++) {
			int diff = x[j] - x[i];
			if (map[diff] == 1) return 0;
			map[diff] = 1;
		}
	}
	return 1;
}

__device__ int recurse(int n, int k, int *x,int *p, int in) {
	int isg = isGolomb(x, in);
	if (!isg) return 0;

	if (in == n-1) {
		if (isGolomb(x, n) && (x[n-1]<p[n-1] || p[n-1]==0)) { for (int i = 0; i < n; i++) p[i] = x[i];return 1; }
		else if(isGolomb(x,n)) return 1;
		else return 0;
	}

	x[in] = x[in - 1] + 1;
	while (x[in] <= k - n + in) {
		int res = recurse(n, k, x,p,in + 1);
		if (res == 1) return 1;
		x[in]++;
	}

	return 0;
}

//algo 5.3
__global__ void kernel(int *n, int *k,int *p, int *res) {
	int i1 = blockIdx.x + 1;
	int i2 = threadIdx.x + 1;
	
	int x[15];
	x[0] = 0;
	x[1] = i1;x[2] = i2;x[*n - 1] = *k;

	if (*n>3) {
		if (i2 <= i1 || i2 >= *k) return;
		int r = recurse(*n, *k, x, p,3);
		if (r == 1) res[0] = 1;
	}
	else if (*n == 3) {
		if (i2 <= i1 || i2 >= *k) return;
		x[1] = i1;
		x[2] = i2;
		if (!isGolomb(x,3)) return;
		else {
			if (x[*n - 1] < p[*n - 1] || p[*n - 1] == 0) for(int i = 0; i < *n; i++) p[i] = x[i];
			res[0] = 1;
		}
	}
	else if (*n == 2) {
		if (!isGolomb(x,2)) return;
		else {
			for (int i = 0; i < *n; i++) p[i] = x[i];
			res[0] = 1;
		}
	}
	else{
		res[0] = 1;
	}
}

int main() {
	int n, r = 0;
	int p[100];
	scanf("%d", &n);
	int *d_res,*d_p,*d_n,*d_k;
	hipMalloc((void **)&d_res, sizeof(int));
	hipMalloc((void **)&d_n, sizeof(int));
	hipMalloc((void **)&d_k, sizeof(int));
	hipMalloc((void **)&d_p, sizeof(int)*n);
	
	int k = (n*(n - 1)) / 2;
	int k2 = (int)((double)(n*n) - 2 * n*pow(n, 0.5) + pow(n, 0.5) - 2);
	if (k2>k) k = k2;

	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
	while(1) {
		hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
		kernel<<<k-n+3,25>>>(d_n,d_k,d_p, d_res);
		hipDeviceSynchronize();
		hipMemcpy(&r, d_res, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(p, d_p, sizeof(int)*n, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		if (r==1) {
			printf("%d  %d  ", n, p[n-1]);
			for(int i = 0;i < n;i++) printf("%d ", p[i]);
			printf("\n");
			break;
		}else k++;
	}

	return 0;
}